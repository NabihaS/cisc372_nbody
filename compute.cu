#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <stdio.h>

// i think CUDA will handle the distribution of kernels after the calls, its just about figuring out the dimensions for the blocks and declaring that
// Q: do i have one grid???


	/*Kernel 2 computes arrays*/
__global__ void computeAccels(vector3* d_accelvalues, vector3* d_Pos, double* d_mass){
	//first compute the pairwise accelerations.  Effect is on the first argument.
	// we're just going to compute the index using our massive 1d array (our matrix, with the computed indices)

	// so in a grid, the whichBlock*Threadsperblock jumps you to the right block, 
	// then you add the thread index to get to the right thread in that block. and this is a 2d grid so u do that for the y's too
	// then need to map it onto the 2d matrix of accels (really a 1d array currently)
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// z dimension?
	// int z=threadIdx.z;
	
	// NOTE. CURRENTLY D_ACCELVALUES IS A 1D ARRAY SO YOU CANT DOUBLE [][] INDEX LIKE ITS A 2D ARRAY. SO ALL CHECKS NEED TO BE ACCORDING
	// Q: the 1d array is why i changed the i check to num SQUARED, bc thats the full range?
	// -- okay so basically, we're not checking i against the whole array, we're seeing if i makes sense, so i may be under num*2 but be out of bounds for a row?
	if (i < NUMENTITIES && j < NUMENTITIES) { // this means that a specific thread in the allocated grids actually correspond w an entity, bc we may have extra blocks
		if (i==j) {
			FILL_VECTOR(d_accelvalues[i*NUMENTITIES+j],0,0,0); // cant use fill vector if u have k
		}
		else{
			vector3 distance;
			for (int k=0;k<3;k++) distance[k]=d_Pos[i][k]-d_Pos[j][k]; // you need to synthreads here
			double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
			double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
			FILL_VECTOR(d_accelvalues[i*NUMENTITIES+j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);// this uses z index
		}
			
		
	}

}

/*Kernel 3 does sums, and also updates values if reduction is not happening */

__global__ void sumAccelsAndUpdate(vector3* d_accelvalues, vector3* d_Pos, vector3* d_Vel){ 
//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
/*
this is NOT a thread doing the smallest component yet 
launch a thread per column and just loop thru each column
OR to optimize launch block

local thing in kernel called sum, calculate sum with single thread, and put that value in the very beginning of accels. OR a temp array 
your final sum will still be a vector3, it will be the aggregate x's,y's, z's of all the objects
if you do __shared__ then all the threads in the block will be able to access the sum and you can do reduction
if you do separate out the sumaccels and update functions then yes u need
--to have an array of vector3* d_sums where you keep track of the sums. OR, you could just overwrite them in the first column of accels

for this one, you only need j to increment, i is static, and youre walking down the column
*/
// does this actually need a start and end
// with our current 1D grid and block settings for this kernel, i could technically just = blockIdx.x
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < NUMENTITIES) {
	vector3 accel_sum={0,0,0};
	for (int j=0;j<NUMENTITIES;j++){
		for (int k=0;k<3;k++)
			accel_sum[k]+=d_accelvalues[i*NUMENTITIES+j][k];
	}

	// Store the result in the accels array??

	// for now, include update here
	//compute the new velocity based on the acceleration and time interval
	//compute the new position based on the velocity and time interval
	for (int k=0;k<3;k++){
		d_Vel[i][k]+=accel_sum[k]*INTERVAL;
		d_Pos[i][k]+=d_Vel[i][k]*INTERVAL;
	}
	
}	

}

/*Kernel 4 uses sums to do update

__global__ void update(vector3** accels, vector3* d_Pos, vector3* d_Vel){
//compute the new velocity based on the acceleration and time interval
//compute the new position based on the velocity and time interval
//Q: is there a built in way to do reductions
	for (i=0;i<NUMENTITIES;i++){

		for (k=0;k<3;k++){
			d_Vel[i][k]+=accels[k]*INTERVAL;
			d_Pos[i][k]+=d_Vel[i][k]*INTERVAL;
		}
	}
}

*/

//compute: Kernel invocations on the GPU
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

	// define grid with x blocks
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // does this need a z? no? bc the threads are the ones that need the z to compute each in the vector?
	dim3 numBlocks((NUMENTITIES+BLOCK_SIZE-1)/BLOCK_SIZE, (NUMENTITIES+BLOCK_SIZE-1)/BLOCK_SIZE);
	
	// grid for second kernel
	dim3 dimBlock2(1,1); // one thread per block
	dim3 numBlocks2(NUMENTITIES,1); // numentities blocks, 1D

	computeAccels<<<numBlocks, dimBlock>>>(d_accelvalues, d_Pos, d_mass);
	sumAccelsAndUpdate<<<numBlocks2, dimBlock2>>>(d_accelvalues, d_Pos, d_Vel);
	// update<<numBlocks, dimBlock<<(d_accelvalues,d_Pos, d_Vel);

	// Q: Do i need hipDeviceSynchronize() anywhere?
	// need to fundamentally understand the relationship btwn compute.cu and nbody.cu files. what is being accessed


}
